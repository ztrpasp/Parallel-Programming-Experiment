
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
using namespace std;
const int X = 1024;
size_t threadsPerBlock;
size_t numberOfBlocks;
//  Initial array
void A_reset(float *A)
{
    for (int i = 0; i < X; i++)
    {
        for (int j = 0; j < i; j++)
            A[i*X+j] = 0;
        A[i*X+i] = 1.0;
        for (int j = i + 1; j < X; j++)
            A[i*X+j] = rand();
    }
    for (int k = 0; k < X; k++)
        for (int i = k + 1; i < X; i++)
            for (int j = 0; j < X; j++)
                A[i*X+j] += A[k*X+j];
}

// serial
void normal(int n, float *A)
{
    for (int k = 0; k < n; k++)
    {
        for (int j = k + 1; j < n; j++)
        {
            A[k*n+j] /= A[k*n+k];
        }
        A[k*n+k] = 1.0;
        for (int i = k + 1; i < n; i++)
        {
            for (int j = k + 1; j < n; j++)
            {
                A[i*n+j] -= A[i*n+k] * A[k*n+j];
            }
            A[i*n+k] = 0;
        }
    }
}
__global__ void division_kernel(float *A, int k, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index+k+1; i < n ; i += stride)
    {
        float element = A[k*n+k];
        float temp = A[k*n+i];
        //请同学们思考，如果分配的总线程数小于 N 应该怎么办？
        A[k*n+i] = (float)temp / element;
    }
    return;
}

__global__ void eliminate_kernel(float *A, int k, int N)
{
    int tx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tx == 0)
        A[k*N+k] = 1;    //对角线元素设为 1
    int row = k + 1 + blockIdx.x; //每个块负责一行
    while (row < N)
    {
        int tid = threadIdx.x;
        while (k + 1 + tid < N)
        {
            int col = k + 1 + tid;
            float temp_1 = A[row*N+col];
            float temp_2 = A[row*N+k];
            float temp_3 = A[k*N+col];
            A[row*N+col] = temp_1 - temp_2 * temp_3;
            tid = tid + blockDim.x;
        }
        __syncthreads(); //块内同步
        if (threadIdx.x == 0)
        {
            A[row*N+k] = 0;
        }
        row += gridDim.x;
    }
    return;
}

//并行算法
void sp(int n, float *A)
{
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    threadsPerBlock = 32;
    numberOfBlocks = 32 * numberOfSMs;

    // size_t size = n * n * sizeof(float);
    // float **A_d;
    // cudaMalloc((void **)&A_d, size);
    // cudaMemcpy(A_d, A, size, cudaMemcpyHostToDevice);

    for (int k = 0; k < n; k++)
    {
        division_kernel<<<numberOfBlocks, threadsPerBlock>>>(A, k, n); //负责除法任务的核函数
        hipDeviceSynchronize();                                 // CPU 与 GPU 之间的同步函数
        // ret = cudaGetLastError();
        // if (ret != cudaSuccess)
        // {
        //     printf("division_kernel failed, %s\n", cudaGetErrorString(ret));
        // }
        eliminate_kernel<<<numberOfBlocks, threadsPerBlock>>>(A, k, n); //负责消去任务的核函数
        hipDeviceSynchronize();
        // ret = cudaGetLastError();
        // if (ret != cudaSuccess)
        // {
        //     printf("eliminate_kernel failed, %s\n", cudaGetErrorString(ret));
        // }
    }

    //cudaMemcpy(A, A_d, size, cudaMemcpyDeviceToHost);

    //cudaFree(A);

}
int main()
{
    
    size_t size = X * X * sizeof(float);
    float *A;
    hipMallocManaged(&A, size);
    
   
    A_reset(A);
        
    sp(X, A);
    hipFree(A);
    cout << "hello" << endl;
    return 0;
}